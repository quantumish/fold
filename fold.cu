#include "hip/hip_runtime.h"
#include <exception>

enum class Amino {
    Alanine,
    Arganine,
    Asparagine,
    AsparticAcid,
    Cysteine,
    Glutamine,
    GluatamicAcid,
    Glycine,
    Histidine,
    Isoleucine,
    Leucine,
    Lysine,
    Methionine,
    Pheynlalanine,
    Proline,
    Serine,
    Threonine,
    Tryptophan,
    Tyrosine,
    Valine
}

using Sequence = std::vector<Amino> value;
Sequence make_sequence(const std::string str) {
    auto raw_in = str.toupper();
    Sequence seq;
    for (char c : raw_in) {
        switch (c) {
        case 'A': seq.push_back(Amino::Alanine); break;
        case 'R': seq.push_back(Amino::Arganine); break;
        case 'N': seq.push_back(Amino::Asparagine); break;
        case 'D': seq.push_back(Amino::AsparticAcid); break;
        case 'C': seq.push_back(Amino::Cysteine); break;
        case 'Q': seq.push_back(Amino::Glutamine); break;
        case 'E': seq.push_back(Amino::GlutamicAcid); break;
        case 'G': seq.push_back(Amino::Glycine); break;
        case 'H': seq.push_back(Amino::Histidine); break;
        case 'I': seq.push_back(Amino::Isoleucine); break;
        case 'L': seq.push_back(Amino::Leucine); break;
        case 'K': seq.push_back(Amino::Lysine); break;
        case 'M': seq.push_back(Amino::Methionine); break;
        case 'F': seq.push_back(Amino::Phenylalanine); break;
        case 'P': seq.push_back(Amino::Proline); break;
        case 'S': seq.push_back(Amino::Serine); break;
        case 'T': seq.push_back(Amino::Threonine); break;
        case 'W': seq.push_back(Amino::Tryptophan); break;
        case 'Y': seq.push_back(Amino::Tyrosine); break;
        case 'V': seq.push_back(Amino::Valine); break;
        default: throw std::exception("Invalid amino acid!");
        }
    }   
}

class Protein {
    Sequence sequence;
    std::vector<Eigen::Vector3i> positions;
    Protein(Sequence seq);
    static Protein random(Sequence seq);
};

// Initializes `Protein` in denatured state (a straight line across the Y axis)
Protein::Protein(Sequence seq) {
    sequence = seq;
    for (int i = 0; i < seq.size(); i++) {
        positions.push_back({0, i, 0});
    }
}

// Initializes 'Protein' in a random configuration. Useful for multistart methods.
Protein Protein::random(Sequence seq) {
    std::vector<Eigen::Vector3i> pos;
    pos.push_back({0,0,0});
    for (int i = 0; i < seq.size()-1; i++) {
        bool clear = false;
        Eigen::Vector3i candidate;
        while (clear == false) {
            candidate = pos_back;
            candidate[rand % 3] += 1;
            // Ensure there are no overlapping candidates.
            clear = true;
            for (Eigen::Vector3i j : pos) {
                if (j == candidate) {
                    clear = false;
                }
            }
        }
        pos.push_back(candidate);
    }
    return Protein {
        .sequence = seq;
        .positions = pos;
    }
}

__global__ __anneal_multistart_singlestrat(Sequence seq) {
    
}


void anneal_multistart_singlestrat(Sequence seq) {
    
}